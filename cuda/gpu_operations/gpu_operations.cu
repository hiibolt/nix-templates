#include "hip/hip_runtime.h"
#include "../complex_operations/complex_operations.h"
#include "gpu_operations.h"

__global__ void set_all ( double* arr, int rows, int cols, double a, double b ) {
    // threadIdx.x = row
    // threadIdx.y = col
    int ind = (threadIdx.x * cols + threadIdx.y) * 2;
    arr[ ind     ] = a;
    arr[ ind + 1 ] = b;
}
__global__ void add_2d_complex ( double* target, double* source, int rows, int cols ) {
    // threadIdx.x = row
    // threadIdx.y = col
    int ind = (threadIdx.x * cols + threadIdx.y) * 2;

    add_complex(
        &target[ind], &target[ind + 1],
         source[ind],  source[ind + 1]
    );
}