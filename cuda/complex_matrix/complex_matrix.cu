#include "../gpu_operations/gpu_operations.h"
#include "complex_matrix.h"
#include <iostream>

ComplexMatrix::ComplexMatrix( int _rows, int _cols ) {
    hipMallocManaged( &state, _rows * _cols * 2 * sizeof(double));

    rows = _rows;
    cols = _cols;

    dim3 threadsPerBlock( rows, cols );
    set_all<<<1, threadsPerBlock>>>( state, rows, cols, 0.0f, 0.0f );

    hipDeviceSynchronize();
}
int ComplexMatrix::get_rows ( ) {
    return rows;
}
int ComplexMatrix::get_cols ( ) {
    return cols;
}
double* ComplexMatrix::get_state ( ) {
    return state;
}
void ComplexMatrix::add ( ComplexMatrix source ) {
    dim3 threadsPerBlock( rows, cols );

    add_2d_complex<<<1, threadsPerBlock>>>( 
        state, source.get_state(), 
        rows, cols
    );

    hipDeviceSynchronize();
}
void ComplexMatrix::print ( ) {
    for ( int i = 0; i < rows; i++ ) {
        for ( int g = 0; g < cols; g++ ) {
            int ind = (i * cols + g) * 2;
            std::cout << '(' << state[ind] << (state[ind + 1] >= 0 ? " + " : " - ") << state[ind + 1] << "i) ";
        }
        std::cout << std::endl;
    }
}
void ComplexMatrix::set_element ( int row, int col, double a, double b ) {
    int ind = (row * cols + col) * 2;
    state[ ind     ] = a;
    state[ ind + 1 ] = b;
}